#include "image_utils.hpp"
#include "Gaussian.cuh"
using namespace std;

int main() {
    Image instance;
    ifstream file("C:\\Users\\Uzair\\OneDrive\\Documents\\Image_Processor\\images\\input\\pepper.ppm", std::ios::binary);
    if (!file) {
        cerr << "Error: Could not open file\n";
        return 1;
    }

    instance = get_imgData(file);

    size_t imageSize = instance.rawdata.size() * sizeof(unsigned char);
    unsigned char* d_data;
    unsigned char* d_outdata;
    unsigned char* d_outdata_Sobel;

    hipMalloc((void**)&d_data, imageSize);
    hipMalloc((void**)&d_outdata, imageSize);
    hipMalloc((void**)&d_outdata_Sobel , imageSize);

    hipMemcpy(d_data, instance.rawdata.data(), imageSize, hipMemcpyHostToDevice);

    int threadspblock = 256;
    int blockspergrid = (instance.rawdata.size() + threadspblock - 1) / threadspblock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    GaussianFilter<<< blockspergrid, threadspblock >>>(d_data, d_outdata, instance.width, instance.height);
    hipDeviceSynchronize();

    SobelFilter << < blockspergrid, threadspblock >> > (d_outdata, d_outdata_Sobel, instance.width, instance.height);
    hipDeviceSynchronize();
    

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU Time: " << milliseconds << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(instance.rawdata.data(), d_outdata, imageSize, hipMemcpyDeviceToHost);
    process_pgm(instance, "C:\\Users\\Uzair\\OneDrive\\Documents\\Image_Processor\\images\\output\\output_Gaussian.pgm");

    hipMemcpy(instance.rawdata.data(), d_outdata_Sobel, imageSize, hipMemcpyDeviceToHost);
    process_pgm(instance, "C:\\Users\\Uzair\\OneDrive\\Documents\\Image_Processor\\images\\output\\output_Gaussian_Sobel.pgm");

    hipFree(d_data);
    hipFree(d_outdata);
    hipFree(d_outdata_Sobel);

    
    return 0;
}