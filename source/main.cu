#include "image_utils.hpp"
#include "Gaussian.cuh"
using namespace std;

int main() {
    Image instance;
    ifstream file("C:\\Users\\Uzair\\OneDrive\\Documents\\Image_Processor\\images\\input\\pepper.ppm", std::ios::binary);
    if (!file) {
        cerr << "Error: Could not open file\n";
        return 1;
    }

    instance = get_imgData(file);

    size_t imageSize = instance.rawdata.size() * sizeof(unsigned char);
    unsigned char* d_data;
    unsigned char* d_outdata;

    hipMalloc((void**)&d_data, imageSize);
    hipMalloc((void**)&d_outdata, imageSize);

    hipMemcpy(d_data, instance.rawdata.data(), imageSize, hipMemcpyHostToDevice);

    int threadspblock = 256;
    int blockspergrid = (instance.rawdata.size() + threadspblock - 1) / threadspblock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    GaussianFilter<<< blockspergrid, threadspblock >>>(d_data, d_outdata, instance.width, instance.height);

    hipDeviceSynchronize();  

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU Time: " << milliseconds << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(instance.rawdata.data(), d_outdata, imageSize, hipMemcpyDeviceToHost);
    hipFree(d_data);
    hipFree(d_outdata);

    process_pgm(instance, "C:\\Users\\Uzair\\OneDrive\\Documents\\Image_Processor\\images\\output\\output_Gaussian.pgm");

    return 0;


}